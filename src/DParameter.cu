#include "hip/hip_runtime.h"
#include "DParameter.cuh"
#include "ChemData.h"
#include "FlameletLib.cuh"
#include <filesystem>
#include <fstream>

cfd::DParameter::DParameter(cfd::Parameter &parameter, Species &species, Reaction *reaction,
                            FlameletLib *flamelet_lib) :
    myid{parameter.get_int("myid")}, dim{parameter.get_int("dimension")},
    inviscid_scheme{parameter.get_int("inviscid_scheme")},
    reconstruction{parameter.get_int("reconstruction")}, limiter{parameter.get_int("limiter")},
    entropy_fix_factor{parameter.get_real("entropy_fix_factor")},
    viscous_scheme{parameter.get_int("viscous_order")}, rans_model{parameter.get_int("RANS_model")},
    turb_implicit{parameter.get_int("turb_implicit")}, n_var{parameter.get_int("n_var")},
    compressibility_correction{parameter.get_int("compressibility_correction")},
    chemSrcMethod{parameter.get_int("chemSrcMethod")}, n_scalar_transported{parameter.get_int("n_scalar_transported")},
    i_fl{parameter.get_int("i_fl")}, i_fl_cv{parameter.get_int("i_fl_cv")}, i_turb_cv{parameter.get_int("i_turb_cv")},
    Pr(parameter.get_real("prandtl_number")), cfl(parameter.get_real("cfl")),
    gradPInDiffusionFlux{parameter.get_bool("gradPInDiffusionFlux")},
    Prt(parameter.get_real("turbulent_prandtl_number")), Sct(parameter.get_real("turbulent_schmidt_number")),
    c_chi{parameter.get_real("c_chi")} {
  const auto &spec = species;
  n_spec = spec.n_spec;
  n_scalar = parameter.get_int("n_scalar");
  if (reaction != nullptr) {
    n_reac = reaction->n_reac;
  }

  // species info
  auto mem_sz = n_spec * sizeof(real);
  hipMalloc(&mw, mem_sz);
  hipMemcpy(mw, spec.mw.data(), mem_sz, hipMemcpyHostToDevice);
  high_temp_coeff.init_with_size(n_spec, 7);
  hipMemcpy(high_temp_coeff.data(), spec.high_temp_coeff.data(), high_temp_coeff.size() * sizeof(real),
             hipMemcpyHostToDevice);
  low_temp_coeff.init_with_size(n_spec, 7);
  hipMemcpy(low_temp_coeff.data(), spec.low_temp_coeff.data(), low_temp_coeff.size() * sizeof(real),
             hipMemcpyHostToDevice);
  hipMalloc(&t_low, mem_sz);
  hipMalloc(&t_mid, mem_sz);
  hipMalloc(&t_high, mem_sz);
  hipMemcpy(t_low, spec.t_low.data(), mem_sz, hipMemcpyHostToDevice);
  hipMemcpy(t_mid, spec.t_mid.data(), mem_sz, hipMemcpyHostToDevice);
  hipMemcpy(t_high, spec.t_high.data(), mem_sz, hipMemcpyHostToDevice);
  hipMalloc(&LJ_potent_inv, mem_sz);
  hipMemcpy(LJ_potent_inv, spec.LJ_potent_inv.data(), mem_sz, hipMemcpyHostToDevice);
  hipMalloc(&vis_coeff, mem_sz);
  hipMemcpy(vis_coeff, spec.vis_coeff.data(), mem_sz, hipMemcpyHostToDevice);
  WjDivWi_to_One4th.init_with_size(n_spec, n_spec);
  hipMemcpy(WjDivWi_to_One4th.data(), spec.WjDivWi_to_One4th.data(), WjDivWi_to_One4th.size() * sizeof(real),
             hipMemcpyHostToDevice);
  sqrt_WiDivWjPl1Mul8.init_with_size(n_spec, n_spec);
  hipMemcpy(sqrt_WiDivWjPl1Mul8.data(), spec.sqrt_WiDivWjPl1Mul8.data(),
             sqrt_WiDivWjPl1Mul8.size() * sizeof(real), hipMemcpyHostToDevice);
  binary_diffusivity_coeff.init_with_size(n_spec, n_spec);
  hipMemcpy(binary_diffusivity_coeff.data(), spec.binary_diffusivity_coeff.data(),
             binary_diffusivity_coeff.size() * sizeof(real), hipMemcpyHostToDevice);
  kb_over_eps_jk.init_with_size(n_spec, n_spec);
  hipMemcpy(kb_over_eps_jk.data(), spec.kb_over_eps_jk.data(),
             kb_over_eps_jk.size() * sizeof(real), hipMemcpyHostToDevice);
  Sc = parameter.get_real("schmidt_number");

  // reactions info
  if (n_reac > 0) {
    hipMalloc(&reac_type, n_reac * sizeof(integer));
    hipMemcpy(reac_type, reaction->label.data(), n_reac * sizeof(integer), hipMemcpyHostToDevice);
    stoi_f.init_with_size(n_reac, n_spec);
    hipMemcpy(stoi_f.data(), reaction->stoi_f.data(), stoi_f.size() * sizeof(integer), hipMemcpyHostToDevice);
    stoi_b.init_with_size(n_reac, n_spec);
    hipMemcpy(stoi_b.data(), reaction->stoi_b.data(), stoi_b.size() * sizeof(integer), hipMemcpyHostToDevice);
    mem_sz = n_reac * sizeof(real);
    hipMalloc(&reac_order, n_reac * sizeof(integer));
    hipMemcpy(reac_order, reaction->order.data(), n_reac * sizeof(integer), hipMemcpyHostToDevice);
    hipMalloc(&A, mem_sz);
    hipMemcpy(A, reaction->A.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&b, mem_sz);
    hipMemcpy(b, reaction->b.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&Ea, mem_sz);
    hipMemcpy(Ea, reaction->Ea.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&A2, mem_sz);
    hipMemcpy(A2, reaction->A2.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&b2, mem_sz);
    hipMemcpy(b2, reaction->b2.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&Ea2, mem_sz);
    hipMemcpy(Ea2, reaction->Ea2.data(), mem_sz, hipMemcpyHostToDevice);
    third_body_coeff.init_with_size(n_reac, n_spec);
    hipMemcpy(third_body_coeff.data(), reaction->third_body_coeff.data(), third_body_coeff.size() * sizeof(real),
               hipMemcpyHostToDevice);
    hipMalloc(&troe_alpha, mem_sz);
    hipMemcpy(troe_alpha, reaction->troe_alpha.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&troe_t3, mem_sz);
    hipMemcpy(troe_t3, reaction->troe_t3.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&troe_t1, mem_sz);
    hipMemcpy(troe_t1, reaction->troe_t1.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&troe_t2, mem_sz);
    hipMemcpy(troe_t2, reaction->troe_t2.data(), mem_sz, hipMemcpyHostToDevice);
  }

  if (flamelet_lib != nullptr) {
    n_z = flamelet_lib->n_z;
    n_zPrime = flamelet_lib->n_zPrime;
    n_chi = flamelet_lib->n_chi;

    mem_sz = (n_z + 1) * sizeof(real);
    hipMalloc(&mix_frac, mem_sz);
    hipMemcpy(mix_frac, flamelet_lib->z.data(), mem_sz, hipMemcpyHostToDevice);
    zPrime.init_with_size(n_zPrime + 1, n_z + 1);
    hipMemcpy(zPrime.data(), flamelet_lib->zPrime.data(), zPrime.size() * sizeof(real), hipMemcpyHostToDevice);
    chi_min.init_with_size(n_zPrime + 1, n_z + 1);
    hipMemcpy(chi_min.data(), flamelet_lib->chi_min.data(), chi_min.size() * sizeof(real), hipMemcpyHostToDevice);
    chi_max.init_with_size(n_zPrime + 1, n_z + 1);
    hipMemcpy(chi_max.data(), flamelet_lib->chi_max.data(), chi_max.size() * sizeof(real), hipMemcpyHostToDevice);
    chi_min_j.init_with_size(n_zPrime + 1, n_z + 1);
    hipMemcpy(chi_min_j.data(), flamelet_lib->chi_min_j.data(), chi_min_j.size() * sizeof(integer),
               hipMemcpyHostToDevice);
    chi_max_j.init_with_size(n_zPrime + 1, n_z + 1);
    hipMemcpy(chi_max_j.data(), flamelet_lib->chi_max_j.data(), chi_max_j.size() * sizeof(integer),
               hipMemcpyHostToDevice);

    chi_ave.allocate_memory(n_chi, n_zPrime + 1, n_z + 1, 0);
    hipMemcpy(chi_ave.data(), flamelet_lib->chi_ave.data(), sizeof(real) * chi_ave.size(), hipMemcpyHostToDevice);
    yk_lib.allocate_memory(n_spec, n_chi, n_zPrime + 1, n_z + 1, 0);
    hipMemcpy(yk_lib.data(), flamelet_lib->yk.data(), sizeof(real) * yk_lib.size() * (n_z + 1),
               hipMemcpyHostToDevice);

    // See if we have computed n_fl_step previously
    if (std::filesystem::exists("output/message/flamelet_step.txt")) {
      std::ifstream fin("output/message/flamelet_step.txt");
      fin >> n_fl_step;
      fin.close();
    } else {
      n_fl_step = 0;
    }
  }

  memset(limit_flow.ll, 0, sizeof(real) * LimitFlow::max_n_var);
  memset(limit_flow.ul, 0, sizeof(real) * LimitFlow::max_n_var);
  memset(limit_flow.sv_inf, 0, sizeof(real) * (MAX_SPEC_NUMBER + 2));
  // density limits
  limit_flow.ll[0] = 1e-6 * parameter.get_real("rho_inf");
  limit_flow.ul[0] = 1e+3 * parameter.get_real("rho_inf");
  for (integer l = 1; l < 4; ++l) {
    // u,v,w
    limit_flow.ll[l] = -1e+3 * parameter.get_real("v_inf");
    limit_flow.ul[l] = 1e+3 * parameter.get_real("v_inf");
  }
  // pressure limits
  limit_flow.ll[4] = 1e-6 * parameter.get_real("p_inf");
  limit_flow.ul[4] = 1e+3 * parameter.get_real("p_inf");
  if (rans_model == 2) {
    // SST model
    limit_flow.ul[5] = std::numeric_limits<real>::max();
    limit_flow.ul[6] = std::numeric_limits<real>::max();
  }
  auto &sv_inf{parameter.get_real_array("sv_inf")};
  for (integer l = 0; l < n_scalar; ++l) {
    limit_flow.sv_inf[l] = sv_inf[l];
  }
}
