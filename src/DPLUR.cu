#include "hip/hip_runtime.h"
#include "DPLUR.cuh"
#include "BoundCond.cuh"

namespace cfd {
__global__ void convert_dq_back_to_dqDt(DZone *zone, const DParameter *param) {
  const integer extent[3]{zone->mx, zone->my, zone->mz};
  const integer i = blockDim.x * blockIdx.x + threadIdx.x;
  const integer j = blockDim.y * blockIdx.y + threadIdx.y;
  const integer k = blockDim.z * blockIdx.z + threadIdx.z;
  if (i >= extent[0] || j >= extent[1] || k >= extent[2]) return;

  const real dt_local = zone->dt_local(i, j, k);
  auto &dq = zone->dq;
  for (integer l = 0; l < param->n_var; ++l) {
    dq(i, j, k, l) /= dt_local;
  }
}

__global__ void set_dq_to_0(const DParameter *param, DZone *zone, integer i_face) {
  const auto &b = zone->boundary[i_face];
  auto range_start = b.range_start, range_end = b.range_end;
  integer i = range_start[0] + (integer) (blockDim.x * blockIdx.x + threadIdx.x);
  integer j = range_start[1] + (integer) (blockDim.y * blockIdx.y + threadIdx.y);
  integer k = range_start[2] + (integer) (blockDim.z * blockIdx.z + threadIdx.z);
  if (i > range_end[0] || j > range_end[1] || k > range_end[2]) return;

  for (integer l = 0; l < param->n_var; ++l) {
    zone->dq(i, j, k, l) = 0;
  }
}

void set_wall_dq_to_0(const Block &block, const DParameter *param, DZone *zone, DBoundCond &bound_cond) {
  for (size_t l = 0; l < bound_cond.n_wall; l++) {
    const auto nb = bound_cond.wall_info[l].n_boundary;
    for (size_t i = 0; i < nb; i++) {
      auto [i_zone, i_face] = bound_cond.wall_info[l].boundary[i];
      if (i_zone != block.block_id) {
        continue;
      }
      const auto &hf = block.boundary[i_face];
      const auto ngg = block.ngg;
      uint tpb[3], bpg[3];
      for (size_t j = 0; j < 3; j++) {
        auto n_point = hf.range_end[j] - hf.range_start[j] + 1;
        tpb[j] = n_point <= (2 * ngg + 1) ? 1 : 16;
        bpg[j] = (n_point - 1) / tpb[j] + 1;
      }
      dim3 TPB{tpb[0], tpb[1], tpb[2]}, BPG{bpg[0], bpg[1], bpg[2]};
      set_dq_to_0<<<BPG, TPB>>>(param, zone, i_face);
    }
  }
}

}