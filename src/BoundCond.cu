#include "hip/hip_runtime.h"
#include "BoundCond.cuh"
#include "Parallel.h"

#ifdef GPU
namespace cfd {

template<typename BCType>
void register_bc(BCType *&bc, int n_bc, std::vector<integer> &indices, BCInfo *&bc_info, Species &species,
                 Parameter &parameter) {
  if (n_bc <= 0) {
    return;
  }

  hipMalloc(&bc, n_bc * sizeof(BCType));
  bc_info = new BCInfo[n_bc];
  for (integer i = 0; i < n_bc; ++i) {
    const integer index = indices[i];
    for (auto &bc_name: parameter.get_string_array("boundary_conditions")) {
      auto &this_bc = parameter.get_struct(bc_name);
      integer bc_label = std::get<integer>(this_bc.at("label"));
      if (index != bc_label) {
        continue;
      }
      bc_info[i].label = bc_label;
      BCType bound_cond(bc_name, parameter);
      hipMemcpy(&(bc[i]), &bound_cond, sizeof(BCType), hipMemcpyHostToDevice);
      break;
    }
  }
//  hipMalloc(&bc, n_bc * sizeof(BCType));
//  bc_info = new BCInfo[n_bc];
//  integer counter = 0;
//  while (counter < n_bc) {
//    BCType bctemp(indices[counter]);
//    bc_info[counter].label = indices[counter];
//    hipMemcpy(&(bc[counter]), &bctemp, sizeof(BCType), hipMemcpyHostToDevice);
//    ++counter;
//  }
}

template<>
void register_bc<Wall>(Wall *&bc, integer n_bc, std::vector<integer> &indices, BCInfo *&bc_info, Species &species,
                       Parameter &parameter) {
  if (n_bc <= 0) {
    return;
  }

  hipMalloc(&bc, n_bc * sizeof(Wall));
  bc_info = new BCInfo[n_bc];
  for (integer i = 0; i < n_bc; ++i) {
    const integer index = indices[i];
    for (auto &bc_name: parameter.get_string_array("boundary_conditions")) {
      auto &this_bc = parameter.get_struct(bc_name);
      integer bc_label = std::get<integer>(this_bc.at("label"));
      if (index != bc_label) {
        continue;
      }
      Wall wall(this_bc);
      bc_info[i].label = bc_label;
      hipMemcpy(&(bc[i]), &wall, sizeof(Wall), hipMemcpyHostToDevice);
    }
  }
}

template<>
void register_bc<Inflow>(Inflow *&bc, integer n_bc, std::vector<integer> &indices, BCInfo *&bc_info, Species &species,
                         Parameter &parameter) {
  if (n_bc <= 0) {
    return;
  }

  hipMalloc(&bc, n_bc * sizeof(Inflow));
  bc_info = new BCInfo[n_bc];
  for (integer i = 0; i < n_bc; ++i) {
    const integer index = indices[i];
    for (auto &bc_name: parameter.get_string_array("boundary_conditions")) {
      auto &this_bc = parameter.get_struct(bc_name);
      integer bc_label = std::get<integer>(this_bc.at("label"));
      if (index != bc_label) {
        continue;
      }
      bc_info[i].label = bc_label;
      Inflow inflow(bc_name, species, parameter);
      inflow.copy_to_gpu(&(bc[i]), species, parameter);
      break;
    }
  }
}

template<>
void
register_bc<FarField>(FarField *&bc, integer n_bc, std::vector<integer> &indices, BCInfo *&bc_info, Species &species,
                      Parameter &parameter) {
  if (n_bc <= 0) {
    return;
  }

  hipMalloc(&bc, n_bc * sizeof(FarField));
  bc_info = new BCInfo[n_bc];
  for (integer i = 0; i < n_bc; ++i) {
    const integer index = indices[i];
    for (auto &bc_name: parameter.get_string_array("boundary_conditions")) {
      auto &this_bc = parameter.get_struct(bc_name);
      integer bc_label = std::get<integer>(this_bc.at("label"));
      if (index != bc_label) {
        continue;
      }
      bc_info[i].label = bc_label;
      FarField farfield(species, parameter);
      farfield.copy_to_gpu(&(bc[i]), species, parameter);
//      hipMemcpy(&(bc[i]), &farfield, sizeof(FarField), hipMemcpyHostToDevice);
      break;
    }
  }
}

template<>
void register_bc<SubsonicInflow>(SubsonicInflow *&bc, integer n_bc, std::vector<integer> &indices,
                                 BCInfo *&bc_info, Species &species, Parameter &parameter) {
  if (n_bc <= 0) {
    return;
  }

  hipMalloc(&bc, n_bc * sizeof(SubsonicInflow));
  bc_info = new BCInfo[n_bc];
  for (integer i = 0; i < n_bc; ++i) {
    const integer index = indices[i];
    for (auto &bc_name: parameter.get_string_array("boundary_conditions")) {
      auto &this_bc = parameter.get_struct(bc_name);
      integer bc_label = std::get<integer>(this_bc.at("label"));
      if (index != bc_label) {
        continue;
      }
      bc_info[i].label = bc_label;
      SubsonicInflow subsonic_inflow(bc_name, parameter);
      subsonic_inflow.copy_to_gpu(&(bc[i]), species, parameter);
      break;
    }
  }
}

template<>
void register_bc<BackPressure>(BackPressure *&bc, integer n_bc, std::vector<integer> &indices,
                               BCInfo *&bc_info, Species &species, Parameter &parameter) {
  if (n_bc <= 0) {
    return;
  }

  hipMalloc(&bc, n_bc * sizeof(BackPressure));
  bc_info = new BCInfo[n_bc];
  for (integer i = 0; i < n_bc; ++i) {
    const integer index = indices[i];
    for (auto &bc_name: parameter.get_string_array("boundary_conditions")) {
      auto &this_bc = parameter.get_struct(bc_name);
      integer bc_label = std::get<integer>(this_bc.at("label"));
      if (index != bc_label) {
        continue;
      }
      bc_info[i].label = bc_label;
      BackPressure back_pressure(bc_name, parameter);
      hipMemcpy(&(bc[i]), &back_pressure, sizeof(BackPressure), hipMemcpyHostToDevice);
      break;
    }
  }
}

void DBoundCond::initialize_bc_on_GPU(Mesh &mesh, std::vector<Field> &field, Species &species, Parameter &parameter) {
  std::vector<integer> bc_labels;
  // Count the number of distinct boundary conditions
  for (auto i = 0; i < mesh.n_block; i++) {
    for (auto &b: mesh[i].boundary) {
      auto lab = b.type_label;
      bool has_this_bc = false;
      for (auto l: bc_labels) {
        if (l == lab) {
          has_this_bc = true;
          break;
        }
      }
      if (!has_this_bc) {
        bc_labels.push_back(lab);
      }
    }
  }
  // Initialize the inflow and wall conditions which are different among cases.
  std::vector<integer> wall_idx, symmetry_idx, inflow_idx, outflow_idx, farfield_idx, subsonic_inflow_idx, back_pressure_idx;
  auto &bcs = parameter.get_string_array("boundary_conditions");
  for (auto &bc_name: bcs) {
    auto &bc = parameter.get_struct(bc_name);
    auto label = std::get<integer>(bc.at("label"));

    auto this_iter = bc_labels.end();
    for (auto iter = bc_labels.begin(); iter != bc_labels.end(); ++iter) {
      if (*iter == label) {
        this_iter = iter;
        break;
      }
    }
    if (this_iter != bc_labels.end()) {
      bc_labels.erase(this_iter);
      auto type = std::get<std::string>(bc.at("type"));
      if (type == "wall") {
        wall_idx.push_back(label);
        ++n_wall;
      } else if (type == "inflow") {
        inflow_idx.push_back(label);
        ++n_inflow;
      }
        // Note: Normally, this would not happen for outflow, symmetry, and periodic conditions.
        // Because the above-mentioned conditions normally do not need to specify special treatments.
        // If we need to add supports for these conditions, then we add them here.
      else if (type == "outflow") {
        outflow_idx.push_back(label);
        ++n_outflow;
      } else if (type == "symmetry") {
        symmetry_idx.push_back(label);
        ++n_symmetry;
      } else if (type == "farfield") {
        farfield_idx.push_back(label);
        ++n_farfield;
      } else if (type == "subsonic_inflow") {
        subsonic_inflow_idx.push_back(label);
        ++n_subsonic_inflow;
      } else if (type == "back_pressure") {
        back_pressure_idx.push_back(label);
        ++n_back_pressure;
      }
    }
  }
  for (int lab: bc_labels) {
    if (lab == 2) {
      wall_idx.push_back(lab);
      ++n_wall;
    } else if (lab == 3) {
      symmetry_idx.push_back(lab);
      ++n_symmetry;
    } else if (lab == 4) {
      farfield_idx.push_back(lab);
      ++n_farfield;
    } else if (lab == 5) {
      inflow_idx.push_back(lab);
      ++n_inflow;
    } else if (lab == 6) {
      outflow_idx.push_back(lab);
      ++n_outflow;
    } else if (lab == 7) {
      subsonic_inflow_idx.push_back(lab);
      ++n_subsonic_inflow;
    } else if (lab == 9) {
      back_pressure_idx.push_back(lab);
      ++n_back_pressure;
    }
  }

  // Read specific conditions
  // We always first initialize the Farfield and Inflow conditions, because they may set the reference values.
  register_bc<FarField>(farfield, n_farfield, farfield_idx, farfield_info, species, parameter);
  register_bc<Inflow>(inflow, n_inflow, inflow_idx, inflow_info, species, parameter);
  register_bc<SubsonicInflow>(subsonic_inflow, n_subsonic_inflow, subsonic_inflow_idx, subsonic_inflow_info, species,
                              parameter);
  register_bc<Wall>(wall, n_wall, wall_idx, wall_info, species, parameter);
  register_bc<Symmetry>(symmetry, n_symmetry, symmetry_idx, symmetry_info, species, parameter);
  register_bc<Outflow>(outflow, n_outflow, outflow_idx, outflow_info, species, parameter);
  register_bc<BackPressure>(back_pressure, n_back_pressure, back_pressure_idx, back_pressure_info, species, parameter);

  link_bc_to_boundaries(mesh, field);

  MpiParallel::barrier();
  if (parameter.get_int("myid") == 0) {
    printf("Finish setting up boundary conditions.\n");
  }
}

void DBoundCond::link_bc_to_boundaries(Mesh &mesh, std::vector<Field> &field) const {
  const integer n_block{mesh.n_block};
  auto **i_wall = new integer *[n_wall];
  for (size_t i = 0; i < n_wall; i++) {
    i_wall[i] = new integer[n_block];
    for (integer j = 0; j < n_block; j++) {
      i_wall[i][j] = 0;
    }
  }
  auto **i_symm = new integer *[n_symmetry];
  for (size_t i = 0; i < n_symmetry; i++) {
    i_symm[i] = new integer[n_block];
    for (integer j = 0; j < n_block; j++) {
      i_symm[i][j] = 0;
    }
  }
  auto **i_farfield = new integer *[n_farfield];
  for (size_t i = 0; i < n_farfield; ++i) {
    i_farfield[i] = new integer[n_block];
    for (integer j = 0; j < n_block; ++j) {
      i_farfield[i][j] = 0;
    }
  }
  auto **i_inflow = new integer *[n_inflow];
  for (size_t i = 0; i < n_inflow; i++) {
    i_inflow[i] = new integer[n_block];
    for (integer j = 0; j < n_block; j++) {
      i_inflow[i][j] = 0;
    }
  }
  auto **i_outflow = new integer *[n_outflow];
  for (size_t i = 0; i < n_outflow; i++) {
    i_outflow[i] = new integer[n_block];
    for (integer j = 0; j < n_block; j++) {
      i_outflow[i][j] = 0;
    }
  }
  auto **i_subsonic_inflow = new integer *[n_subsonic_inflow];
  for (size_t i = 0; i < n_subsonic_inflow; i++) {
    i_subsonic_inflow[i] = new integer[n_block];
    for (integer j = 0; j < n_block; j++) {
      i_subsonic_inflow[i][j] = 0;
    }
  }
  auto **i_back_pressure = new integer *[n_back_pressure];
  for (size_t i = 0; i < n_back_pressure; i++) {
    i_back_pressure[i] = new integer[n_block];
    for (integer j = 0; j < n_block; j++) {
      i_back_pressure[i][j] = 0;
    }
  }

  // We first count how many faces corresponds to a given boundary condition
  for (integer i = 0; i < n_block; i++) {
    count_boundary_of_type_bc(mesh[i].boundary, n_wall, i_wall, i, n_block, wall_info);
    count_boundary_of_type_bc(mesh[i].boundary, n_symmetry, i_symm, i, n_block, symmetry_info);
    count_boundary_of_type_bc(mesh[i].boundary, n_farfield, i_farfield, i, n_block, farfield_info);
    count_boundary_of_type_bc(mesh[i].boundary, n_inflow, i_inflow, i, n_block, inflow_info);
    count_boundary_of_type_bc(mesh[i].boundary, n_outflow, i_outflow, i, n_block, outflow_info);
    count_boundary_of_type_bc(mesh[i].boundary, n_subsonic_inflow, i_subsonic_inflow, i, n_block,
                              subsonic_inflow_info);
    count_boundary_of_type_bc(mesh[i].boundary, n_back_pressure, i_back_pressure, i, n_block, back_pressure_info);
  }
  for (size_t l = 0; l < n_wall; l++) {
    wall_info[l].boundary = new int2[wall_info[l].n_boundary];
  }
  for (size_t l = 0; l < n_symmetry; ++l) {
    symmetry_info[l].boundary = new int2[symmetry_info[l].n_boundary];
  }
  for (size_t l = 0; l < n_farfield; ++l) {
    farfield_info[l].boundary = new int2[farfield_info[l].n_boundary];
  }
  for (size_t l = 0; l < n_inflow; l++) {
    inflow_info[l].boundary = new int2[inflow_info[l].n_boundary];
  }
  for (size_t l = 0; l < n_outflow; l++) {
    outflow_info[l].boundary = new int2[outflow_info[l].n_boundary];
  }
  for (size_t l = 0; l < n_subsonic_inflow; ++l) {
    subsonic_inflow_info[l].boundary = new int2[subsonic_inflow_info[l].n_boundary];
  }
  for (size_t l = 0; l < n_back_pressure; ++l) {
    back_pressure_info[l].boundary = new int2[back_pressure_info[l].n_boundary];
  }

  const auto ngg{mesh[0].ngg};
  for (auto i = 0; i < n_block; i++) {
    link_boundary_and_condition(mesh[i].boundary, wall_info, n_wall, i_wall, i);
    link_boundary_and_condition(mesh[i].boundary, symmetry_info, n_symmetry, i_symm, i);
    link_boundary_and_condition(mesh[i].boundary, farfield_info, n_farfield, i_farfield, i);
    link_boundary_and_condition(mesh[i].boundary, inflow_info, n_inflow, i_inflow, i);
    link_boundary_and_condition(mesh[i].boundary, outflow_info, n_outflow, i_outflow, i);
    link_boundary_and_condition(mesh[i].boundary, subsonic_inflow_info, n_subsonic_inflow, i_subsonic_inflow, i);
    link_boundary_and_condition(mesh[i].boundary, back_pressure_info, n_back_pressure, i_back_pressure, i);
  }
  for (auto i = 0; i < n_block; i++) {
    for (size_t l = 0; l < n_wall; l++) {
      const auto nb = wall_info[l].n_boundary;
      for (size_t m = 0; m < nb; m++) {
        auto i_zone = wall_info[l].boundary[m].x;
        if (i_zone != i) {
          continue;
        }
        auto &b = mesh[i].boundary[wall_info[l].boundary[m].y];
        for (int q = 0; q < 3; ++q) {
          if (q == b.face) continue;
          b.range_start[q] += ngg;
          b.range_end[q] -= ngg;
        }
      }
    }
    hipMemcpy(field[i].h_ptr->boundary, mesh[i].boundary.data(), mesh[i].boundary.size() * sizeof(Boundary),
               hipMemcpyHostToDevice);
  }
  for (integer i = 0; i < n_wall; i++) {
    delete[]i_wall[i];
  }
  for (integer i = 0; i < n_symmetry; i++) {
    delete[]i_symm[i];
  }
  for (integer i = 0; i < n_farfield; ++i) {
    delete[]i_farfield[i];
  }
  for (integer i = 0; i < n_inflow; i++) {
    delete[]i_inflow[i];
  }
  for (integer i = 0; i < n_outflow; i++) {
    delete[]i_outflow[i];
  }
  for (integer i = 0; i < n_subsonic_inflow; ++i) {
    delete[]i_subsonic_inflow[i];
  }
  for (integer i = 0; i < n_back_pressure; ++i) {
    delete[]i_back_pressure[i];
  }
  delete[]i_wall;
  delete[]i_symm;
  delete[]i_farfield;
  delete[]i_inflow;
  delete[]i_outflow;
  delete[]i_subsonic_inflow;
  delete[]i_back_pressure;
}

void count_boundary_of_type_bc(const std::vector<Boundary> &boundary, integer n_bc, integer **sep, integer blk_idx,
                               integer n_block, BCInfo *bc_info) {
  if (n_bc <= 0) {
    return;
  }

  // Count how many faces correspond to the given bc
  const auto n_boundary{boundary.size()};
  auto *n = new integer[n_bc];
  memset(n, 0, sizeof(integer) * n_bc);
  for (size_t l = 0; l < n_bc; l++) {
    integer label = bc_info[l].label; // This means every bc should have a member "label"
    for (size_t i = 0; i < n_boundary; i++) {
      auto &b = boundary[i];
      if (b.type_label == label) {
        ++bc_info[l].n_boundary;
        ++n[l];
      }
    }
  }
  if (blk_idx < n_block - 1) {
    for (size_t l = 0; l < n_bc; l++) {
      sep[l][blk_idx + 1] = n[l] + sep[l][blk_idx];
    }
  }
  delete[]n;
}

void link_boundary_and_condition(const std::vector<Boundary> &boundary, BCInfo *bc, integer n_bc, integer **sep,
                                 integer i_zone) {
  const auto n_boundary{boundary.size()};
  for (size_t l = 0; l < n_bc; l++) {
    integer label = bc[l].label;
    int has_read{sep[l][i_zone]};
    for (auto i = 0; i < n_boundary; i++) {
      auto &b = boundary[i];
      if (b.type_label == label) {
        bc[l].boundary[has_read] = make_int2(i_zone, i);
        ++has_read;
      }
    }
  }
}

void Inflow::copy_to_gpu(Inflow *d_inflow, Species &spec, const Parameter &parameter) {
  const integer n_scalar{parameter.get_int("n_scalar")};
  real *h_sv = new real[n_scalar];
  for (integer l = 0; l < n_scalar; ++l) {
    h_sv[l] = sv[l];
  }
  delete[]sv;
  hipMalloc(&sv, n_scalar * sizeof(real));
  hipMemcpy(sv, h_sv, n_scalar * sizeof(real), hipMemcpyHostToDevice);

  hipMemcpy(d_inflow, this, sizeof(Inflow), hipMemcpyHostToDevice);
}

void FarField::copy_to_gpu(FarField *d_farfield, Species &spec, const Parameter &parameter) {
  const integer n_scalar{parameter.get_int("n_scalar")};
  real *h_sv = new real[n_scalar];
  for (integer l = 0; l < n_scalar; ++l) {
    h_sv[l] = sv[l];
  }
  delete[]sv;
  hipMalloc(&sv, n_scalar * sizeof(real));
  hipMemcpy(sv, h_sv, n_scalar * sizeof(real), hipMemcpyHostToDevice);

  hipMemcpy(d_farfield, this, sizeof(FarField), hipMemcpyHostToDevice);
}

void SubsonicInflow::copy_to_gpu(cfd::SubsonicInflow *d_inflow, cfd::Species &spec, const cfd::Parameter &parameter) {
  const integer n_scalar{parameter.get_int("n_scalar")};
  real *h_sv = new real[n_scalar];
  for (integer l = 0; l < n_scalar; ++l) {
    h_sv[l] = sv[l];
  }
  delete[]sv;
  hipMalloc(&sv, n_scalar * sizeof(real));
  hipMemcpy(sv, h_sv, n_scalar * sizeof(real), hipMemcpyHostToDevice);

  hipMemcpy(d_inflow, this, sizeof(SubsonicInflow), hipMemcpyHostToDevice);
}
} // cfd
#endif
